#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h> // 添加half精度支持
#include "radixSelectTopK_thrust.cuh"

// 比较函数用于验证结果
bool greater_than(half a, half b)
{
    return __half2float(a) > __half2float(b); // 转换为float进行比较
}

// 用于排序的pair结构
struct ValueIndexPair {
    half value;
    int index;
    
    bool operator<(const ValueIndexPair& other) const {
        return __half2float(value) > __half2float(other.value); // 降序排列，需要转换为float比较
    }
};

// 验证结果是否正确
bool validateResults(half* h_output, uint* h_indices, ValueIndexPair* h_reference_pairs, half* h_input, int k) {
    bool success = true;
    
    // 创建临时数组来储存GPU结果的值-索引对，用于排序
    ValueIndexPair *gpu_pairs = new ValueIndexPair[k];
    for (int i = 0; i < k; i++) {
        gpu_pairs[i].value = h_output[i];
        gpu_pairs[i].index = h_indices[i];
    }
    
    // 排序GPU结果
    std::sort(gpu_pairs, gpu_pairs + k);
    
    // 验证值和索引
    for (int i = 0; i < k; i++) {
        // 验证值是否正确 (使用更大的容差，因为FP16精度较低)
        if (std::abs(__half2float(gpu_pairs[i].value) - __half2float(h_reference_pairs[i].value)) > 1e-2) {
            printf("错误: 位置 %d, GPU值: %f, CPU值: %f\n", 
                   i, __half2float(gpu_pairs[i].value), __half2float(h_reference_pairs[i].value));
            success = false;
            break;
        }
        
        // 验证索引是否正确（通过检查该索引在原始数组中的值是否等于当前值）
        if (std::abs(__half2float(h_input[gpu_pairs[i].index]) - __half2float(gpu_pairs[i].value)) > 1e-2) {
            printf("索引错误: 位置 %d, 索引 %d, 索引对应的原始值: %f, 期望值: %f\n", 
                   i, gpu_pairs[i].index, __half2float(h_input[gpu_pairs[i].index]), 
                   __half2float(gpu_pairs[i].value));
            success = false;
            break;
        }
    }
    
    delete[] gpu_pairs;
    return success;
}

// 测试radixSelectTopKThrustK方法
float testRadixSelectTopKThrust(half* h_input, half* h_output, uint* h_indices, 
                        half* d_input, half* d_output, uint* d_indices, 
                        ValueIndexPair* h_reference_pairs, 
                        int num_items, int k, int run, bool validate) {
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // 将数据复制到设备
    hipMemcpy(d_input, h_input, num_items * sizeof(half), hipMemcpyHostToDevice);

    // 计时开始
    hipDeviceSynchronize();
    hipEventRecord(start);
    
    // 执行TopK
    hipError_t error = radixSelectTopKThrust<half>(d_input, num_items, k, d_output, d_indices);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // 等待GPU完成
    hipDeviceSynchronize();
    if (error != hipSuccess) {
        printf("CUDA错误: %s\n", hipGetErrorString(error));
        return -1.0f;
    }
    
    float kernel_milliseconds = 0;
    hipEventElapsedTime(&kernel_milliseconds, start, stop);
    
    hipMemcpy(h_output, d_output, k * sizeof(half), hipMemcpyDeviceToHost);
    hipMemcpy(h_indices, d_indices, k * sizeof(uint), hipMemcpyDeviceToHost);

    if (run % 10 == 0) {
        printf("radixSelectTopK GPU计算的前10个值: ");
        for (int i = 0; i < std::min(10, k); i++) {
            printf("%f(索引%d) ", __half2float(h_output[i]), h_indices[i]);
        }
        printf("\n");
    }

    // 验证结果
    if (validate) {
        bool success = validateResults(h_output, h_indices, h_reference_pairs, h_input, k);
        if (success) {
            printf("验证成功! radixSelectTopK 结果与索引均正确\n");
        } else {
            printf("验证失败! radixSelectTopK 结果或索引不正确\n");
        }
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return kernel_milliseconds;
}

// 测试radixSelectTopKPreallocated方法
float testRadixSelectTopKPreallocatedThrust(half* h_input, half* h_output, uint* h_indices, 
                                    half* d_input, half* d_output, uint* d_indices, 
                                    ValueIndexPair* h_reference_pairs, 
                                    int num_items, int k, int run, bool validate) {
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // 预分配内存
    half* d_keys_sorted = nullptr;
    uint* d_indices_in = nullptr;
    hipMalloc(&d_keys_sorted, num_items * sizeof(half));
    hipMalloc(&d_indices_in, num_items * sizeof(uint));
    
    // 将数据复制到设备
    hipMemcpy(d_input, h_input, num_items * sizeof(half), hipMemcpyHostToDevice);

    // 计时开始
    hipDeviceSynchronize();
    hipEventRecord(start);
    
    // 执行TopK
    hipError_t error = radixSelectTopKPreallocatedThrust<half>(
        d_input, num_items, k, d_output, d_indices, d_keys_sorted, d_indices_in);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // 等待GPU完成
    hipDeviceSynchronize();
    if (error != hipSuccess) {
        printf("CUDA错误: %s\n", hipGetErrorString(error));
        hipFree(d_keys_sorted);
        hipFree(d_indices_in);
        return -1.0f;
    }
    
    float kernel_milliseconds = 0;
    hipEventElapsedTime(&kernel_milliseconds, start, stop);
    
    hipMemcpy(h_output, d_output, k * sizeof(half), hipMemcpyDeviceToHost);
    hipMemcpy(h_indices, d_indices, k * sizeof(uint), hipMemcpyDeviceToHost);

    if (run % 10 == 0) {
        printf("radixSelectTopKPreallocated GPU计算的前10个值: ");
        for (int i = 0; i < std::min(10, k); i++) {
            printf("%f(索引%d) ", __half2float(h_output[i]), h_indices[i]);
        }
        printf("\n");
    }

    // 验证结果
    if (validate) {
        bool success = validateResults(h_output, h_indices, h_reference_pairs, h_input, k);
        if (success) {
            printf("验证成功! radixSelectTopKPreallocated 结果与索引均正确\n");
        } else {
            printf("验证失败! radixSelectTopKPreallocated 结果或索引不正确\n");
        }
    }
    
    // 释放预分配的内存
    hipFree(d_keys_sorted);
    hipFree(d_indices_in);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return kernel_milliseconds;
}

int main()
{
    // 测试参数
    const int num_items = 21600;
    const int k = 1024;
    const int num_runs = 100;
    
    // 选择是否验证结果（每10次运行验证一次）
    const bool validate = true;

    // 分配主机内存
    half *h_input = new half[num_items];
    half *h_output = new half[k];
    uint *h_indices = new uint[k];
    ValueIndexPair *h_reference_pairs = new ValueIndexPair[num_items]; // 用于验证的参考结果

    // 分配设备内存
    half *d_input, *d_output;
    uint *d_indices;
    hipMalloc(&d_input, num_items * sizeof(half));
    hipMalloc(&d_output, k * sizeof(half));
    hipMalloc(&d_indices, k * sizeof(uint));

    // 性能记录
    float total_time_standard = 0.0f;
    float total_time_preallocated = 0.0f;
    
    printf("开始测试 radixSelectTopK 和 radixSelectTopKPreallocated...\n");
    printf("参数: 元素总数 = %d, k = %d, 运行次数 = %d\n", num_items, k, num_runs);
    
    // 生成随机数据
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
    
    for (int run = 0; run < num_runs; run++)
    {
        if (run % 10 == 0) {
            printf("\n运行 %d/%d\n", run+1, num_runs);
            printf("生成随机数据...\n");
        }
        
        // 生成随机数据
        for (int i = 0; i < num_items; i++)
        {
            float rand_val = dist(gen);
            h_input[i] = __float2half(rand_val); // 转换float到half
            
            // 确保有一些已知的大值进行测试
            if (i < 10) {
                h_input[i] = __float2half(0.9f + i * 0.01f);  // 前10个元素为0.90, 0.91, 0.92...0.99
            }
            
            h_reference_pairs[i].value = h_input[i];
            h_reference_pairs[i].index = i;
        }

        // 只在每10次运行时打印数据集统计信息
        if (run % 10 == 0) {
            half min_val = h_input[0], max_val = h_input[0];
            float sum_val = 0;
            for (int i = 0; i < num_items; i++) {
                if (__half2float(h_input[i]) < __half2float(min_val))
                    min_val = h_input[i];
                if (__half2float(h_input[i]) > __half2float(max_val))
                    max_val = h_input[i];
                sum_val += __half2float(h_input[i]);
            }
            printf("数据集统计: 最小值=%f, 最大值=%f, 平均值=%f\n", 
                   __half2float(min_val), __half2float(max_val), sum_val/num_items);
        }

        // 在CPU上计算参考结果
        std::sort(h_reference_pairs, h_reference_pairs + num_items);

        if (run % 10 == 0) {
            printf("CPU排序后的前10个值: ");
            for (int i = 0; i < std::min(10, k); i++) {
                printf("%f ", __half2float(h_reference_pairs[i].value));
            }
            printf("\n");
        }

        // 测试标准版本
        float time_standard = testRadixSelectTopKThrust(h_input, h_output, h_indices, 
                                               d_input, d_output, d_indices, 
                                               h_reference_pairs, num_items, k, run, 
                                               validate && (run % 10 == 0));
        total_time_standard += time_standard;
        
        // 测试预分配版本
        float time_preallocated = testRadixSelectTopKPreallocatedThrust(h_input, h_output, h_indices, 
                                                              d_input, d_output, d_indices, 
                                                              h_reference_pairs, num_items, k, run, 
                                                              validate && (run % 10 == 0));
        total_time_preallocated += time_preallocated;
        
        if (run % 10 == 0) {
            printf("运行 %d: radixSelectTopK 用时: %.3f ms, radixSelectTopKPreallocated 用时: %.3f ms\n", 
                   run+1, time_standard, time_preallocated);
        }
    }
    
    // 输出平均性能结果
    float avg_time_standard = total_time_standard / num_runs;
    float avg_time_preallocated = total_time_preallocated / num_runs;
    
    printf("\n性能总结 (%d 次运行):\n", num_runs);
    printf("radixSelectTopKThrust 平均用时: %.3f ms\n", avg_time_standard);
    printf("radixSelectTopKPreallocatedThrust 平均用时: %.3f ms\n", avg_time_preallocated);
    printf("性能提升: %.2f%%\n", 100.0f * (avg_time_standard - avg_time_preallocated) / avg_time_standard);

    // 释放内存
    delete[] h_input;
    delete[] h_output;
    delete[] h_indices;
    delete[] h_reference_pairs;
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_indices);

    return 0;
}