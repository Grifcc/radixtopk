#include "hip/hip_runtime.h"
#include "radixSelectTopK_thrust.cuh"
#include "utils.cuh"

// 模板函数实现
template<typename KeyT>
hipError_t radixSelectTopKThrust(
    KeyT *d_keys_in, 
    unsigned int num_items, 
    unsigned int k, 
    KeyT *d_keys_out, 
    unsigned int *d_indices_out, 
    hipStream_t stream) {
  
  hipError_t error = hipSuccess;
  
  // 分配临时存储
  KeyT *d_keys_sorted = nullptr;
  unsigned int *d_indices_in = nullptr;
  
  hipMalloc(&d_keys_sorted, sizeof(KeyT) * num_items);
  hipMalloc(&d_indices_in, sizeof(unsigned int) * num_items);
  
  // 初始化索引数组
  unsigned int block_size = 256;
  unsigned int grid_size = (num_items + block_size - 1) / block_size;
  radixtopk::init_indices_kernel<<<grid_size, block_size, 0, stream>>>(d_indices_in, num_items);
  
  // 复制输入键到排序数组
  hipMemcpyAsync(d_keys_sorted, d_keys_in, sizeof(KeyT) * num_items, 
                 hipMemcpyDeviceToDevice, stream);
  
  // 使用Thrust进行排序（降序）
  thrust::sort_by_key(thrust::cuda::par.on(stream), 
                     d_keys_sorted, d_keys_sorted + num_items,
                     d_indices_in,
                     thrust::greater<KeyT>());
  
  // 复制Top-K结果
  hipMemcpyAsync(d_keys_out, d_keys_sorted, sizeof(KeyT) * k, 
                 hipMemcpyDeviceToDevice, stream);
  hipMemcpyAsync(d_indices_out, d_indices_in, sizeof(unsigned int) * k, 
                 hipMemcpyDeviceToDevice, stream);
  
  // 释放临时内存
  hipFree(d_keys_sorted);
  hipFree(d_indices_in);
  
  return error;
}

template<typename KeyT>
hipError_t radixSelectTopKPreallocatedThrust(
    const KeyT *d_keys_in, 
    unsigned int num_items, 
    unsigned int k, 
    KeyT *d_keys_out, 
    unsigned int *d_indices_out,
    KeyT *d_keys_sorted, 
    unsigned int *d_indices_in,
    hipStream_t stream) {
  
  hipError_t error = hipSuccess;
  
  // 初始化索引数组
  unsigned int block_size = 256;
  unsigned int grid_size = (num_items + block_size - 1) / block_size;
  radixtopk::init_indices_kernel<<<grid_size, block_size, 0, stream>>>(d_indices_in, num_items);
  
  // 复制输入键到排序数组
  hipMemcpyAsync(d_keys_sorted, d_keys_in, sizeof(KeyT) * num_items, 
                 hipMemcpyDeviceToDevice, stream);
  
  // 使用Thrust进行排序（降序）
  thrust::sort_by_key(thrust::cuda::par.on(stream), 
                     d_keys_sorted, d_keys_sorted + num_items,
                     d_indices_in,
                     thrust::greater<KeyT>());
  
  // 复制Top-K结果
  hipMemcpyAsync(d_keys_out, d_keys_sorted, sizeof(KeyT) * k, 
                 hipMemcpyDeviceToDevice, stream);
  hipMemcpyAsync(d_indices_out, d_indices_in, sizeof(unsigned int) * k, 
                 hipMemcpyDeviceToDevice, stream);
  
  return error;
}

// 显式实例化模板函数，确保编译器生成所需的代码
template hipError_t radixSelectTopKThrust<float>(
    float*, unsigned int, unsigned int, 
    float*, unsigned int*, hipStream_t);

template hipError_t radixSelectTopKThrust<__half>(
    __half*, unsigned int, unsigned int, 
    __half*, unsigned int*, hipStream_t);

template hipError_t radixSelectTopKPreallocatedThrust<float>(
    const float*, unsigned int, unsigned int, 
    float*, unsigned int*, float*, unsigned int*, hipStream_t);

template hipError_t radixSelectTopKPreallocatedThrust<__half>(
    const __half*, unsigned int, unsigned int, 
    __half*, unsigned int*, __half*, unsigned int*, hipStream_t);
